/**
  * Maestría en Ciencias - Mención Informática
  * -------------------------------------------
  * Escriba un programa CUDA que calcule C = n*A + B, en donde A, B, C son vectores
  * y n una constante escalar.
  *
  * Adaptado de https://www.olcf.ornl.gov/tutorials/cuda-vector-addition/
  * 
  * Presentado por:
  * Zuñiga Rojas, Gabriela
  * Soncco Pimentel, Braulio
  */
  
#include <hip/hip_runtime.h>
#include <stdio.h>
  #include <stdlib.h>
  #include <math.h>

  hipEvent_t start, stop;
  float elapsedTime;

  const int k = 5;
  
  // CUDA kernel. Each thread takes care of one element of c
  __global__ void vecAdd(double *a, double *b, double *c, int n, int k)
  {

      // Get our global thread ID
      int id = blockIdx.x*blockDim.x+threadIdx.x;
   
      // Make sure we do not go out of bounds
      if (id < n)
          c[id] = k * a[id] + b[id];
    
  }
   
  int main( int argc, char* argv[] )
  {
  
      // Size of vectors
      int n = 10000000;
   
      // Host input vectors
      double *h_a;
      double *h_b;
      //Host output vector
      double *h_c;
   
      // Device input vectors
      double *d_a;
      double *d_b;
      //Device output vector
      double *d_c;
   
      // Size, in bytes, of each vector
      size_t bytes = n*sizeof(double);
   
      // Allocate memory for each vector on host
      h_a = (double*)malloc(bytes);
      h_b = (double*)malloc(bytes);
      h_c = (double*)malloc(bytes);
   
      // Allocate memory for each vector on GPU
      hipMalloc(&d_a, bytes);
      hipMalloc(&d_b, bytes);
      hipMalloc(&d_c, bytes);
   
      int i;
      // Initialize vectors on host
      for( i = 0; i < n; i++ ) {
          h_a[i] = sin(i)*sin(i);
          h_b[i] = cos(i)*cos(i);
      }
   
      // Copy host vectors to device
      hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
      hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
   
      int blockSize, gridSize;
   
      // Number of threads in each thread block
      blockSize = 1024;
   
      // Number of thread blocks in grid
      gridSize = (int)ceil((float)n/blockSize);

      hipEventCreate(&start);
      hipEventRecord(start);
   
      // Execute the kernel
      vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n, k);

    hipEventCreate(&stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
   
      // Copy array back to host
      hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
   
      // Sum up vector c and print result divided by n, this should equal 1 within error
      double sum = 0;
      for(i=0; i<n; i++)
          sum += h_c[i];
      printf("final result: %f\n", sum/n);
   
      // Release device memory
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
   
      // Release host memory
      free(h_a);
      free(h_b);
      free(h_c);
  
        

      hipEventElapsedTime(&elapsedTime, start,stop);
      printf("%f milisegundos\n" ,elapsedTime);
   
      return 0;
  }